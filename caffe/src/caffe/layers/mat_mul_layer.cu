#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/mat_mul_layer.hpp"

namespace caffe {

template <typename Dtype>
void MatMulLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data_0 = bottom[0]->gpu_data();  // blob
  const Dtype* bottom_data_1 = bottom[1]->gpu_data();  // matrix
  Dtype* top_data = top[0]->mutable_gpu_data();

  vector<int> bottom_0_shape = bottom[0]->shape();
  vector<int> bottom_1_shape = bottom[1]->shape();

  const int num = bottom_0_shape[0];
  const int m_ = bottom_0_shape[1] * bottom_0_shape[2];
  const int k_ = bottom_0_shape[3];
  const int n_ = bottom_1_shape[2];

  for (int i = 0; i < num; ++i) {
    for (int j = 0; j < num_kernels_; ++j) {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, m_, n_, k_, (Dtype)1.,
          bottom_data_0 + bottom[0]->offset(i),
          bottom_data_1 + bottom[1]->offset(i, j),
          (Dtype)0., top_data + top[0]->offset(i, channels_ * j));
    }
  }
}

template <typename Dtype>
void MatMulLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {

    const Dtype* top_diff = top[0]->gpu_diff();

    vector<int> top_0_shape = top[0]->shape();
    vector<int> bottom_0_shape = bottom[0]->shape();
    vector<int> bottom_1_shape = bottom[1]->shape();

    const int num = bottom_0_shape[0];
    // const int chan = bottom_0_shape[1];

    // this should NOT be done since it would destroy gradients
    // of other layer, zeroing the diff is done by the solver
    caffe_gpu_set(bottom[0]->count(), (Dtype)0., bottom[0]->mutable_gpu_diff());

    for (int i = 0; i < num; ++i) {
      for (int j = 0; j < num_kernels_; ++j) {
        caffe_gpu_gemm<Dtype>(CblasNoTrans,
                              CblasNoTrans,
                              top[0]->shape()[2] * bottom[0]->shape()[1],
                              bottom[0]->shape()[3],
                              top[0]->shape()[3],
                              (Dtype)1.,
                              top_diff + top[0]->offset(i, channels_ * j),
                              bottom[1]->gpu_data() + bottom[1]->offset(i, j),
                              (Dtype)1.,
                              bottom[0]->mutable_gpu_diff() +
                              bottom[0]->offset(i));
      }
    }
  }
  if (propagate_down[1]) {
    // THIS HAS TO BE CHECKED, SOMEHOW IT SEEMS WEIRD THAT WE SET THIS TO
    // ZERO, WE MIGHT OVERWRITE ALL THE OTHER DIFFS
    // IT IS HOWEVER REQUIRED TO PASS THE GRADIENT TESTS
    // this should NOT be done since it would destroy gradients
    // of other layer, zeroing the diff is done by the solver
    caffe_gpu_set(bottom[1]->count(), (Dtype)0., bottom[1]->mutable_gpu_diff());
    const Dtype* top_diff = top[0]->gpu_diff();

    vector<int> top_0_shape = top[0]->shape();
    vector<int> bottom_0_shape = bottom[0]->shape();
    vector<int> bottom_1_shape = bottom[1]->shape();

    const int num = bottom_0_shape[0];

    for (int i = 0; i < num; ++i) {
      for (int j = 0; j < num_kernels_; ++j) {
        caffe_gpu_gemm<Dtype>(CblasTrans,
                              CblasNoTrans,
                              top[0]->shape()[3],
                              bottom[0]->shape()[3],
                              bottom[0]->shape()[2]*bottom[0]->shape()[1],
                              (Dtype)1.,
                              top_diff + top[0]->offset(i, channels_ * j),
                              bottom[0]->gpu_data() + bottom[0]->offset(i),
                              (Dtype)1.,
                              bottom[1]->mutable_gpu_diff() +
                              bottom[1]->offset(i, j));
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(MatMulLayer);

}  // namespace caffe
